/*********************************************************************
Author: Gabriel Hofer
Date: October 19, 2020
Instructor: Dr. Karlsson
Course: CSC-410 Parallel Computing
*********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define THREADS_PER_BLOCK 512
const int inf = 32768;

void printA(int * A, const int n);

/*********************************************************************
Floyd helper (auxiliary)
*********************************************************************/
__global__ void aux(int * dA, const int n, const int k){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index > n*n) return;
	__syncthreads();
  int i = index / n, j = index % n;
  dA[i*n+j] = dA[i*n+j] < (dA[i*n+k]+dA[k*n+j]) ? dA[i*n+j] : dA[i*n+k]+dA[k*n+j];
}

/*********************************************************************
Floyd-Warshall Algorithm
*********************************************************************/
void floyd(int * dA, const int n){
	for(int k=0;k<n;k++){
		aux<<<(n*n+THREADS_PER_BLOCK)/(THREADS_PER_BLOCK),THREADS_PER_BLOCK>>>(dA,n,k);
		hipDeviceSynchronize();
	}
}

/*********************************************************************
Print Array
*********************************************************************/
void printA(int * A, const int n){
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			printf("%d\t",A[i*n+j]);
			hipDeviceSynchronize();
		}
		printf("\n");
		hipDeviceSynchronize();
	}
	printf("\n");
	hipDeviceSynchronize();
}

/*********************************************************************
Serial, used for checking correctness 
*********************************************************************/
void serial(int * A, const int n){
	printA(A,n);
	for(int k=0;k<n;k++)
		for(int i=0;i<n;i++)
			for(int j=0;j<n;j++)
				A[i*n+j] = A[i*n+j] < (A[i*n+k]+A[k*n+j]) ? A[i*n+j] : A[i*n+k]+A[k*n+j];
	printA(A,n);
}

/*********************************************************************
Usage Statement
*********************************************************************/
void Usage(){ printf("Usage: ./cuda -N n_integer\n"); }

/*********************************************************************
Make Random Matrix 
*********************************************************************/
int * makeMatrix(const int n){
	int * A = (int *)malloc(n*n*sizeof(int));
	srand(time(0));
	for(int i=0;i<n;i++)
		for(int j=0;j<n;j++){
			if(rand()&1) A[i*n+j]=(rand()%20)+1; // random number in range [1,20]
			else A[i*n+j]=inf;
		}
	return A;
}

/*********************************************************************
Main
*********************************************************************/
int main(int argc, char *argv[]) {
	if(argc==1) {
		Usage();
		return 0;
	}
	// convert argument to integer
  int n = atoi(argv[2]);

	// allocate memory for graph
	int * A = makeMatrix(n);
 
	// size of A in bytes
  int Asize = n*n*sizeof(int);

	// print before
	// printA(A,n);

  // allocate 2D array on Device
  int * dA=NULL;
  hipMalloc((void **)&dA, Asize);

  // copy Array to Device
  hipMemcpy(dA, A, Asize, hipMemcpyHostToDevice);

	clock_t before = clock();

	// run the algorithm
  floyd(dA,n);

	clock_t after = clock();
	printf("Execution Time: %f\n", (float)(after-before)/CLOCKS_PER_SEC);

  // copy Array from Device to Host
  hipMemcpy(A, dA, Asize, hipMemcpyDeviceToHost);

	// print result
	// printA(A,n);

  // Cleanup
  hipFree(dA);
  hipDeviceSynchronize();
	
	//serial(n);
  return 0;
}


