#include "hip/hip_runtime.h"
/*********************************************************************
Author: Gabriel Hofer
Date: October 19, 2020
Instructor: Dr. Karlsson
Course: CSC-410 Parallel Computing
*********************************************************************/
#include <stdio.h>
#define THREADS_PER_BLOCK 512

void printA(int * A, const int n);

/*********************************************************************
Parallel stuff, Floyd helper
*********************************************************************/
__global__ void aux(int * dA, const int n, const int k){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index > n*n) return;
	__syncthreads();
  int i = index / n;
	int j = index % n;
  dA[i*n+j] = dA[i*n+j] < (dA[i*n+k]+dA[k*n+j]) ? dA[i*n+j] : dA[i*n+k]+dA[k*n+j];
}

/*********************************************************************
Floyd-Warshall Algorithm
*********************************************************************/
void floyd(const int n){
  // size of A in bytes
  int Asize = n*n*sizeof(int);
	int inf = 512;
  // allocate 2D array on Host
  int * A = (int *)malloc(Asize);

	int tmp[n*n] = { 
		0, 2, 5, inf, inf, inf, 
		inf, 0, 7, 1, inf, 8, 
		inf, inf, 0, 4, inf, inf, 
		inf, inf, inf, 0, 3, inf, 
		inf, inf, 2, inf, 0, 3, 
		inf, 5, inf, 2, 4, 0 };

	memcpy(A,tmp,n*n*sizeof(int));

	// print before
	printA(A,n);

  // allocate 2D array on Device
  int * dA=NULL;
  hipMalloc((void **)&dA, Asize);

  // copy Array to Device
  hipMemcpy(dA, A, Asize, hipMemcpyHostToDevice);

  for(int k=0;k<n;k++){
    // call floyd's algorithm
    aux<<<(n*n+THREADS_PER_BLOCK)/(THREADS_PER_BLOCK),THREADS_PER_BLOCK>>>(dA,n,k);
		hipDeviceSynchronize();
  }

  // copy Array back to Host
  hipMemcpy(A, dA, Asize, hipMemcpyDeviceToHost);

	// print result
	printA(A,n);

  // Cleanup
  hipFree(dA);
  hipDeviceSynchronize();
}

/*********************************************************************
Print Array
*********************************************************************/
void printA(int * A, const int n){
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			printf("%d\t",A[i*n+j]);
			hipDeviceSynchronize();
		}
		printf("\n");
		hipDeviceSynchronize();
	}
	printf("\n");
	hipDeviceSynchronize();
}

/*********************************************************************
Serial
*********************************************************************/
void serial(const int n){
	int Asize = n*n*sizeof(int);
	int inf = 512;
  // allocate 2D array on Host
  int * A = (int *)malloc(Asize);

	int tmp[n*n] = { 
		0, 2, 5, inf, inf, inf, 
		inf, 0, 7, 1, inf, 8, 
		inf, inf, 0, 4, inf, inf, 
		inf, inf, inf, 0, 3, inf, 
		inf, inf, 2, inf, 0, 3, 
		inf, 5, inf, 2, 4, 0 };

	memcpy(A,tmp,n*n*sizeof(int));

	// print before
	printA(A,n);
	for(int k=0;k<n;k++)
		for(int i=0;i<n;i++)
			for(int j=0;j<n;j++)
				A[i*n+j] = A[i*n+j] < (A[i*n+k]+A[k*n+j]) ? A[i*n+j] : A[i*n+k]+A[k*n+j];
	printA(A,n);
}

/*********************************************************************
Main
*********************************************************************/
int main() {
  int n;
	printf("size of array: ");
  hipDeviceSynchronize();
  scanf("%d", &n);
  hipDeviceSynchronize();
	printf("%d\n", n);
  hipDeviceSynchronize();

  floyd(n);
	//serial(n);
  return 0;
}


